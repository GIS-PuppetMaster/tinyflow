#include "hip/hip_runtime.h"
#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <>
#include <algorithm>
//��
#include <hipDNN.h>
#include <stdlib.h>
#include <iostream>
//��-
using namespace std;
#define MAX_THREADS_NUM 512
#define MAX_BLOCKS_NUM 4096
#define BLOCK_NUM(count) min(((count + MAX_THREADS_NUM - 1) / MAX_THREADS_NUM), MAX_BLOCKS_NUM)
#define CUDA_1D_KERNEL_LOOP(i, n) \
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
        i += blockDim.x * gridDim.x)

//��
#define CUDNN_CALL(f) { \
hipdnnStatus_t err = (f); \
if (err != HIPDNN_STATUS_SUCCESS) {\
    \
        std::cout << "    Error occurred: " << err << std::endl; \
        std::exit(1); \
} \
}
//��-
__global__ void matrix_array_set_kernel(int count,
                                        float *arr,
                                        float value) {
  CUDA_1D_KERNEL_LOOP(index, count) {
    arr[index] = value;
  }
}

__global__ void matrix_broadcast_to_kernel(int inputCount, float* inputArr,
                                           int outputCount, float* outputArr) {
  CUDA_1D_KERNEL_LOOP(index, outputCount) {
      outputArr[index] = inputArr[index % inputCount];
  }
}

__global__ void matrix_reduce_sum_axis_zero_kernel(float* inputArr,
                                                   int outputCount, float* outputArr,
                                                   int zeroDim) {
      CUDA_1D_KERNEL_LOOP(index, outputCount) {
          float sum = 0;
          for (int i = 0; i < zeroDim; ++i) {
              sum += inputArr[index + i * outputCount];
          }
          outputArr[index] = sum;
      }
}



__global__ void matrix_elementwise_add_kernel(float* matAData, float* matBData,
                                              float* outputData, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputData[index] = matAData[index] + matBData[index];
    }
}

__global__ void matrix_elementwise_add_by_const_kernel(float* inputArr, float val,
                                                       float* outputArr, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = inputArr[index] + val;
    }
}

__global__ void matrix_elementwise_multiply_kernel(float* matAData, float* matBData,
                                                   float* outputData, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputData[index] = matAData[index] * matBData[index];
    }
}

__global__ void matrix_elementwise_multipy_by_const_kernel(float* inputArr, float val,
                                                           float* outputArr, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = inputArr[index] * val;
    }
}

__global__ void matrix_relu_kernel(float* inputArr, float* outputArr, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = inputArr[index];
        if (inputArr[index] < 0) {
            outputArr[index] = 0.f;
        }
    }
}

__global__ void matrix_relu_gradient_kernel(const float* inputArr, const float* gradArr,
                                            float* outputArr, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = inputArr[index] > 0 ? gradArr[index] : 0;
    }
}

__global__ void matrix_softmax_kernel(int nRow, int nCol, float* inputArr, float* outputArr) {
    int y = blockIdx.x * blockDim.x + threadIdx.x;
    if (y >= nRow) return;

    float* input = inputArr + y * nCol;
    float* output = outputArr + y * nCol;

    float maxval = *input;
    for (int i = 1; i < nCol; ++i) {
        maxval = max(input[i], maxval);
    }
    float sum = 0;
    for (int i = 0; i < nCol; ++i) {
        sum += expf(input[i] - maxval);
    }
    for (int i = 0; i < nCol; ++i) {
        output[i] = expf(input[i] - maxval) / sum;
    }
}

/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)
__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  input_b += y * ncol;
  float maxval = *input_a;
  // Find max for a row.
  for (int x = 1; x < ncol; ++x) {
    maxval = max(maxval, input_a[x]);
  }
  // Deduct by max for a row, and raise to exp.
  float sum = 0;
  for (int x = 0; x < ncol; ++x) {
    sum += exp(input_a[x] - maxval);
  }
  // Compute per-row loss.
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
    loss -= input_b[x] * log(exp(input_a[x] - maxval) / sum);
  }
  loss_per_row[y] = loss;
  __syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }
    mean_loss /= nrow;
    output[0] = mean_loss;
  }
}

__global__ void matrix_exp_kernel(float* inputArr, float* outputArr, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = exp(inputArr[index]);
    }
}

__global__ void matrix_log_kernel(float* inputArr, float* outputArr, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = log(inputArr[index]);
    }
}

__global__ void matrix_reverse_kernel(float* inputArr, float* outputArr, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = 1. / inputArr[index];
    }
}

__global__ void matrix_pow_kernel(float* inputArr, float val, float* outputArr, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = pow(inputArr[index],val);
    }
}




int DLGpuArraySet(DLArrayHandle arr, float value) {
  int count = 1;
  for (int i = 0; i < arr->ndim; ++i) {
    count *= arr->shape[i];
  }
  float *arr_data = (float *)arr->data;
  matrix_array_set_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
    count, arr_data, value);
  return 0;
}

int DLGpuBroadcastTo(const DLArrayHandle input, DLArrayHandle output) {
  assert(input->ndim + 1 == output->ndim);
  int inputCount = 1, outputCount = output->shape[0];
  for (int i = 0; i < input->ndim; ++i) {
      assert(input->shape[i] == output->shape[i + 1]);
      inputCount *= input->shape[i];
      outputCount *= output->shape[i + 1];
  }
  float* inputArr = (float*) input->data;
  float* outputArr = (float*) output->data;
  matrix_broadcast_to_kernel<<<BLOCK_NUM(outputCount), MAX_THREADS_NUM>>>(
    inputCount, inputArr, outputCount, outputArr);
  return 0;
}


__global__ void matrix_reduce_sum_axis_n_kernel(float* inputArr,
                                                int outputCount, float* outputArr,
                                                int reduceDim,int stride,int lowstride) {
    CUDA_1D_KERNEL_LOOP(index, outputCount) {
        int lown = index / stride;
        int lown1 = index % stride;
        float sum = 0;
        for (int i = 0; i < reduceDim; ++i) {
        sum += inputArr[lown * lowstride + lown1 + i * stride];
        }
        outputArr[index] = sum;
    }
}

__global__ void matrix_reduce_sum_axis_n_kernel_backward(float* inputArr,
                                                        int outputCount, float* outputArr,
                                                        int reduceDim,int lowstride) {
    CUDA_1D_KERNEL_LOOP(index, outputCount) {

        int lown = index / lowstride / reduceDim;
        int lown1 = index % lowstride;
        outputArr[index] += inputArr[lown * lowstride + lown1];

    }
}



int DLGpuReduceSumAxisN(const DLArrayHandle input, DLArrayHandle output, const int axis) {


    if(input->ndim == 1){
    assert(1 == output->ndim);
    }else{
    assert(input->ndim == output->ndim + 1);
    }

    int stride = 1;

    for (int i = input->ndim; i > axis + 1; --i) {

        stride = stride * (input->shape[i-1]);
    }
    int reduceDim = input->shape[axis], outputCount = 1;
    for (int i = 0; i < output->ndim; ++i) {

        if( i < axis){
            assert(input->shape[i] == output->shape[i]);
        }else if(input->ndim != 1){
            assert(input->shape[i+1] == output->shape[i]);
        }

        outputCount *= output->shape[i];
    }
    float* inputArr = (float*) input->data;
    float* outputArr = (float*) output->data;
   // printf("%d",reduceDim);
   // printf("%d",outputCount);
   // printf("%d",stride);
    int lowstride = reduceDim * stride;
    matrix_reduce_sum_axis_n_kernel<<<BLOCK_NUM(outputCount), MAX_THREADS_NUM>>>(
            inputArr, outputCount, outputArr, reduceDim, stride, lowstride);

    return 0;
}

int DLGpuReduceSumAxisNBackward(const DLArrayHandle input, DLArrayHandle output, const int axis) {


    if(output->ndim == 1){
        assert(1 == input->ndim);
    }else{
        assert(input->ndim +1== output->ndim);
    }

    int lowstride = 1;

    for (int i = (output->ndim) - 1; i > axis; --i) {
        lowstride = lowstride * (output->shape[i]);
    }

    int reduceDim = output->shape[axis], outputCount = 1;
    for (int i = 0; i < output->ndim; ++i) {



        outputCount *= output->shape[i];
    }
    float* inputArr = (float*) input->data;
    float* outputArr = (float*) output->data;
    matrix_reduce_sum_axis_n_kernel_backward<<<BLOCK_NUM(outputCount), MAX_THREADS_NUM>>>(
        inputArr, outputCount, outputArr, reduceDim, lowstride);

    return 0;
}



int DLGpuReduceSumAll(const DLArrayHandle input, DLArrayHandle output) {

    assert(1 == output->ndim);
    assert(1 == output->shape[0]);
    int stride = 1;
    int reduceDim = input->shape[0];
    int outputCount = 1;
    int lowstride = 1;
    for (int i = 0; i < input->ndim; ++i) {
        outputCount *= input->shape[i];
    }
    float* inputArr = (float*) input->data;
    float* outputArr = (float*) output->data;

    float* tmp = (float*)malloc(sizeof(float)*outputCount);
    float* tmp1 = (float*)malloc(sizeof(float)*outputCount);
    float* inputi = inputArr;
    float* outputi = tmp;
    float* changetmp;
    for(int i= (input->ndim)-1; i > 0;--i){
        outputCount /= input->shape[i];
        reduceDim = input->shape[i];
        lowstride = reduceDim * stride;

        matrix_reduce_sum_axis_n_kernel<<<BLOCK_NUM(outputCount), MAX_THREADS_NUM>>>(
            inputi, outputCount, outputi, reduceDim, stride, lowstride);

        if (i == (input->ndim)-1){
        changetmp = tmp1;
        }else{
        changetmp = inputi;
        }
        inputi = outputi;
        outputi = changetmp;
        
    }

    outputi = outputArr;
    lowstride = reduceDim * stride;
    matrix_reduce_sum_axis_n_kernel<<<BLOCK_NUM(outputCount), MAX_THREADS_NUM>>>(
        inputi, outputCount, outputi, reduceDim, stride, lowstride);
    free(tmp);
    free(tmp1);

    return 0;
}

int DLGpuReduceSumAllBackward(const DLArrayHandle input, DLArrayHandle output) {

    assert(1 == input->ndim);
    assert(1 == input->shape[0]);
    float *val = (float*)malloc(sizeof(float));
    float* inputArr = (float*) input->data;
    hipMemcpy(val, inputArr, sizeof(float), hipMemcpyDeviceToHost);


    DLGpuArraySet(output, *val);
    return 0;
}




int DLGpuReduceSumAxisZero(const DLArrayHandle input, DLArrayHandle output) {
  assert(input->ndim == output->ndim + 1);
  int zeroDim = input->shape[0], outputCount = 1;
    for (int i = 0; i < output->ndim; ++i) {
        assert(input->shape[i+1] == output->shape[i]);
        outputCount *= output->shape[i];
    }
  float* inputArr = (float*) input->data;
  float* outputArr = (float*) output->data;
  matrix_reduce_sum_axis_zero_kernel<<<BLOCK_NUM(outputCount), MAX_THREADS_NUM>>>(
          inputArr, outputCount, outputArr, zeroDim);
  return 0;
}

int DLGpuMatrixElementwiseAdd(const DLArrayHandle matA,
                              const DLArrayHandle matB, DLArrayHandle output) {
  assert(matA->ndim == output->ndim);
  assert(matB->ndim == output->ndim);
  int count = 1;
  for (int i = 0; i < matA->ndim; ++i) {
    assert(matA->shape[i] == output->shape[i]);
    assert(matB->shape[i] == output->shape[i]);
    count *= matA->shape[i];
  }
  float* matAData = (float*) matA->data;
  float* matBData = (float*) matB->data;
  float* outputData = (float*) output->data;
  matrix_elementwise_add_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
          matAData, matBData, outputData, count);
  return 0;
}

int DLGpuMatrixElementwiseAddByConst(const DLArrayHandle input, float val,
                                     DLArrayHandle output) {
  assert(input->ndim == output->ndim);
  int count = 1;
  for (int i = 0; i < input->ndim; ++i) {
    assert(input->shape[i] == output->shape[i]);
    count *= input->shape[i];
  }


  float* inputArr = (float*) input->data;
  float* outputArr = (float*) output->data;
  matrix_elementwise_add_by_const_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
          inputArr, val, outputArr, count);
  return 0;
}

int DLGpuMatrixElementwiseMultiply(const DLArrayHandle matA,
                                   const DLArrayHandle matB,
                                   DLArrayHandle output) {
  assert(matA->ndim == output->ndim);
  assert(matB->ndim == output->ndim);
  int count = 1;
  for (int i = 0; i < matA->ndim; ++i) {
    assert(matA->shape[i] == output->shape[i]);
    assert(matB->shape[i] == output->shape[i]);
    count *= matA->shape[i];
  }
  float* matAData = (float*) matA->data;
  float* matBData = (float*) matB->data;
  float* outputData = (float*) output->data;
  matrix_elementwise_multiply_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
          matAData, matBData, outputData, count);
  return 0;
}

int DLGpuMatrixMultiplyByConst(const DLArrayHandle input, float val,
                               DLArrayHandle output) {
  assert(input->ndim == output->ndim);
  int count = 1;
  for (int i = 0; i < input->ndim; ++i) {
    assert(input->shape[i] == output->shape[i]);

    count *= input->shape[i];
  }


  float* inputArr = (float*) input->data;
  float* outputArr = (float*) output->data;
  matrix_elementwise_multipy_by_const_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
          inputArr, val, outputArr, count);
  return 0;
}

int DLGpuMatrixMultiply(const DLArrayHandle matA, bool transposeA,
                        const DLArrayHandle matB, bool transposeB,
                        DLArrayHandle matC) {
  // Hint: use cublas
  // cublas assume matrix is column major
  assert(matA->ndim == 2);
  assert(matB->ndim == 2);
  assert(matC->ndim == 2);
  assert(matA->shape[transposeA ? 0 : 1] == matB->shape[transposeB ? 1 : 0]);
  assert(matA->shape[transposeA ? 1 : 0] == matC->shape[0]);
  assert(matB->shape[transposeB ? 0 : 1] == matC->shape[1]);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  const float* matAData = (const float*) matA->data;
  const float* matBData = (const float*) matB->data;
  float* matCData = (float*) matC->data;
  float alpha = 1, beta = 0;

  hipblasSgemm(handle,
              (transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
              (transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N),
              (transposeB ? matB->shape[0] : matB->shape[1]),
              (transposeA ? matA->shape[1] : matA->shape[0]),
              (transposeB ? matB->shape[1] : matB->shape[0]),
              &alpha,
              matBData, matB->shape[1],
matAData, matA->shape[1],
& beta,
matCData, (transposeB ? matB->shape[0] : matB->shape[1]));

return 0;
}

int DLGpuRelu(const DLArrayHandle input, DLArrayHandle output) {
    assert(input->ndim == output->ndim);
    int count = 1;
    for (int i = 0; i < input->ndim; ++i) {
        assert(input->shape[i] == output->shape[i]);
        count *= input->shape[i];
    }
    float* inputArr = (float*)input->data;
    float* outputArr = (float*)output->data;
    matrix_relu_kernel << <BLOCK_NUM(count), MAX_THREADS_NUM >> > (
        inputArr, outputArr, count);
    return 0;
}

int DLGpuReluGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
    DLArrayHandle output) {
    assert(input->ndim == in_grad->ndim);
    assert(input->ndim == output->ndim);
    int count = 1;
    for (int i = 0; i < input->ndim; ++i) {
        assert(input->shape[i] == in_grad->shape[i]);
        assert(input->shape[i] == output->shape[i]);
        count *= input->shape[i];
    }
    const float* inputArr = (const float*)input->data;
    const float* gradArr = (const float*)in_grad->data;
    float* outputArr = (float*)output->data;
    matrix_relu_gradient_kernel << <BLOCK_NUM(count), MAX_THREADS_NUM >> > (
        inputArr, gradArr, outputArr, count);
    return 0;
}

int DLGpuSoftmax(const DLArrayHandle input, DLArrayHandle output) {
    assert(input->ndim == 2);
    assert(output->ndim == 2);
    assert(input->shape[0] == output->shape[0]);
    assert(input->shape[1] == output->shape[1]);

    int nRow = input->shape[0];
    int nCol = input->shape[1];

    dim3 block(MAX_THREADS_NUM);
    dim3 grid((nRow + block.x - 1) / block.x);

    float* inputArr = (float*)input->data;
    float* outputArr = (float*)output->data;

    matrix_softmax_kernel << <grid, block >> > (nRow, nCol, inputArr, outputArr);

    return 0;
}

int DLGpuSoftmaxCrossEntropy(const DLArrayHandle input_a,
    const DLArrayHandle input_b,
    DLArrayHandle output) {
    assert(input_a->ndim == 2);
    assert(input_b->ndim == 2);
    assert(output->ndim == 1);
    assert(input_a->shape[0] == input_b->shape[0] &&
        input_a->shape[1] == input_b->shape[1]);
    int nrow = input_a->shape[0];
    // Maximum x- or y-dimension of a block = 1024
    // But we need 'nrow' shared memory, and max shared memory is 48KB.
    // Conservatively allow max 16KB shared memory.
    assert(nrow <= 1024 * 4);
    int ncol = input_a->shape[1];
    const float* input_data_a = (const float*)input_a->data;
    const float* input_data_b = (const float*)input_b->data;
    float* output_data = (float*)output->data;
    dim3 threads;
    if (nrow <= 1024) {
        threads.x = nrow;
    }
    else {
        threads.x = 1024;
        threads.y = (nrow + 1023) / 1024;
    }
    // 1 block, each block with 'threads' number of threads with 'nrow' shared
    // memory size
    matrix_softmax_cross_entropy_kernel << <1, threads, nrow * sizeof(float) >> > (
        nrow, ncol, input_data_a, input_data_b, output_data);
    return 0;
}

int DLGpuMatrixExp(const DLArrayHandle input, DLArrayHandle output) {
    assert(input->ndim == output->ndim);
    int count = 1;
    for (int i = 0; i < input->ndim; ++i) {
    assert(input->shape[i] == output->shape[i]);
    count *= input->shape[i];
    }
    float* inputArr = (float*) input->data;
    float* outputArr = (float*) output->data;
    matrix_exp_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
    inputArr, outputArr, count);
    return 0;
}


int DLGpuMatrixLog(const DLArrayHandle input, DLArrayHandle output) {
    assert(input->ndim == output->ndim);
    int count = 1;
    for (int i = 0; i < input->ndim; ++i) {
    assert(input->shape[i] == output->shape[i]);
    count *= input->shape[i];
    }
    float* inputArr = (float*) input->data;
    float* outputArr = (float*) output->data;
    matrix_log_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
    inputArr, outputArr, count);
    return 0;
}


int DLGpuMatrixReverse(const DLArrayHandle input, DLArrayHandle output) {
    assert(input->ndim == output->ndim);
    int count = 1;
    for (int i = 0; i < input->ndim; ++i) {
    assert(input->shape[i] == output->shape[i]);
    count *= input->shape[i];
    }
    float* inputArr = (float*) input->data;
    float* outputArr = (float*) output->data;
    matrix_reverse_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
    inputArr, outputArr, count);
    return 0;
}

int DLGpuMatrixPow(const DLArrayHandle input,const float val, DLArrayHandle output) {
    assert(input->ndim == output->ndim);
    int count = 1;
    for (int i = 0; i < input->ndim; ++i) {
    assert(input->shape[i] == output->shape[i]);
    count *= input->shape[i];
    }
    float* inputArr = (float*) input->data;
    float* outputArr = (float*) output->data;
    matrix_pow_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
    inputArr, val, outputArr, count);
    return 0;
}




//3ά
int DLGpuConvolution1DForward(const DLArrayHandle input,
    const DLArrayHandle filter,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int v         /*filter stride */) {

    //cout<<dataformat<<endl;
   // cout<<padding<<endl;
    assert(input->ndim == 3);
    assert(filter->ndim == 3);



    int input_n = input->shape[0];
    int input_c = input->shape[2];
    int input_h = 1;
    int input_w = input->shape[1];

    int filter_n = filter->shape[0];
    int filter_c = filter->shape[2];
    int filter_h = 1;
    int filter_w = filter->shape[1];

    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = 1;
        input_w = input->shape[2];

        filter_n = filter->shape[0];
        filter_c = filter->shape[1];
        filter_h = 1;
        filter_w = filter->shape[2];
    }

    int out_n;
    int out_c;
    int out_h;
    int out_w;

    int pad_h = 0;
    int pad_w = 0;

    int u = 1;

    if (padding == 1) {
        pad_w = filter_w / 2;
    }


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));


    //������
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        filter_n,
        filter_c,
        filter_h,
        filter_w));


    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_descriptor,
        pad_h, pad_w, // zero-padding
        u, v, // stride
        1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //����output��4��ά��
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w));

    if (dataformat == 0) {
        assert(output->shape[0] == out_n);
        assert(output->shape[1] == out_c);
        assert(1 == out_h);
        assert(output->shape[2] == out_w);
    }
    else {
        assert(output->shape[0] == out_n);
        assert(output->shape[2] == out_c);
        assert(1 == out_h);
        assert(output->shape[1] == out_w);
    }



    //output��Ϣ
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        out_n,
        out_c,
        out_h,
        out_w));

    //�������㷨
    hipdnnConvolutionFwdAlgo_t algo;
    CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(handle,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        0,
        &algo));

    //׼����������Ŀռ�
    size_t workspace_size = 0;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(handle,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        algo,
        &workspace_size));
    void* workspace = nullptr;
    hipMalloc(&workspace, workspace_size);


    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionForward(handle,
        &alpha, //x*w����
        input_descriptor,
        input->data,
        filter_descriptor,
        filter->data,
        conv_descriptor,
        algo,
        workspace,
        workspace_size,
        &beta, //y����,y�������ݽ������ţ�
        output_descriptor,
        output->data));
    //�ڴ�
    hipFree(workspace);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));

    return 0;

}


int DLGpuConvolution1DForwardGetOutShape(const int* input_shapes,
    const int* filter_shapes,
    int* output_shapes,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int v          /*filter stride */) {

   // cout << dataformat << endl;
   // cout << padding << endl;


    int input_n = input_shapes[0];
    int input_c = input_shapes[2];
    int input_h = 1;
    int input_w = input_shapes[1];


    int filter_n = filter_shapes[0];
    int filter_c = filter_shapes[2];
    int filter_h = 1;
    int filter_w = filter_shapes[1];

    if (dataformat == 0) {
        input_n = input_shapes[0];
        input_c = input_shapes[1];
        input_h = 1;
        input_w = input_shapes[2];

        filter_n = filter_shapes[0];
        filter_c = filter_shapes[1];
        filter_h = 1;
        filter_w = filter_shapes[2];
    }


    int out_n;
    int out_c;
    int out_h;
    int out_w;

    int pad_h = 0;
    int pad_w = 0;

    int u = 1;

    if (padding == 1) {
        pad_w = filter_w / 2;
    }


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));


    //������
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        filter_n,
        filter_c,
        filter_h,
        filter_w));


    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_descriptor,
        pad_h, pad_w, // zero-padding
        u, v, // stride
        1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //����output��4��ά��
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w));



    if (dataformat == 0) {
        output_shapes[0] = out_n;
        output_shapes[1] = out_c;
        output_shapes[2] = out_w;
    }
    else {
        output_shapes[0] = out_n;
        output_shapes[1] = out_w;
        output_shapes[2] = out_c;
    }



    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));

    return 0;

}


int DLGpuConvolution1DBackward(const DLArrayHandle input,
    const DLArrayHandle doutput,
    const DLArrayHandle filter,
    DLArrayHandle dfilter,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int v          /*filter stride */) {

    assert(input->ndim == 3);
    assert(filter->ndim == 3);
        

     int input_n = input->shape[0];
     int input_c = input->shape[2];
     int input_h = 1;
     int input_w = input->shape[1];

     int filter_n = filter->shape[0];
     int filter_c = filter->shape[2];
     int filter_h = 1;
     int filter_w = filter->shape[1];


    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = 1;
        input_w = input->shape[2];

        filter_n = filter->shape[0];
        filter_c = filter->shape[1];
        filter_h = 1;
        filter_w = filter->shape[2];
    }



    int out_n;
    int out_c;
    int out_h;
    int out_w;

    int pad_h = 0;
    int pad_w = 0;

    int u = 1;

    if (padding == 1) {
        pad_w = filter_w / 2;
    }


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));


    //������
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        filter_n,
        filter_c,
        filter_h,
        filter_w));


    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_descriptor,
        pad_h, pad_w, // zero-padding
        u, v, // stride
        1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //����output��4��ά��
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w));

  

    if (dataformat == 0) {
        assert(doutput->shape[0] == out_n);
        assert(doutput->shape[1] == out_c);
        assert(1 == out_h);
        assert(doutput->shape[2] == out_w);
    }
    else {
        assert(doutput->shape[0] == out_n);
        assert(doutput->shape[2] == out_c);
        assert(1 == out_h);
        assert(doutput->shape[1] == out_w);
    }


    //output��Ϣ
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        out_n,
        out_c,
        out_h,
        out_w));

    

    
    //�������㷨
    hipdnnConvolutionBwdFilterAlgo_t  algo1;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterAlgorithm(handle,
        input_descriptor,
        output_descriptor,
        conv_descriptor,
        filter_descriptor,
        HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
        0,
        &algo1));

    hipdnnConvolutionBwdDataAlgo_t algo2;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataAlgorithm(handle,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
        0,
        &algo2));

    //׼����������Ŀռ�


    size_t workspace_size1= 0;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterWorkspaceSize(handle,
        input_descriptor,
        output_descriptor,
        conv_descriptor,
        filter_descriptor,
        algo1,
        &workspace_size1));
    void* workspace1= nullptr;
    hipMalloc(&workspace1, workspace_size1);

    size_t workspace_size2 = 0;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(handle,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        algo2,
        &workspace_size2));
    void* workspace2 = nullptr;
    hipMalloc(&workspace2, workspace_size2);


    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionBackwardFilter(handle,
        &alpha, //x*w����
        input_descriptor,
        input->data,
        output_descriptor,
        doutput->data,
        conv_descriptor,
        algo1,
        workspace1,
        workspace_size1,
        &beta, //y����,y�������ݽ������ţ�
        filter_descriptor,
        dfilter->data));


    CUDNN_CALL(hipdnnConvolutionBackwardData(handle,
        &alpha, //x*w����
        filter_descriptor,
        filter->data,
        output_descriptor,
        doutput->data,
        conv_descriptor,
        algo2,
        workspace2,
        workspace_size2,
        &beta, //y����,y�������ݽ������ţ�
        input_descriptor,
        dinput->data));


    //�ڴ�
    hipFree(workspace1);
    hipFree(workspace2);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));

    return 0;

}





//4ά
int DLGpuConvolution2DForward(const DLArrayHandle input,
    const DLArrayHandle filter,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int u,          /* vertical filter stride */
    const int v          /* horizontal filter stride */){

    assert(input->ndim == 4);
    assert(filter->ndim == 4);


    int input_n = input->shape[0];
    int input_c = input->shape[3];
    int input_h = input->shape[1];
    int input_w = input->shape[2];

    int filter_n = filter->shape[0];
    int filter_c = filter->shape[3];
    int filter_h = filter->shape[1];
    int filter_w = filter->shape[2];


    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = input->shape[2];
        input_w = input->shape[3];

        filter_n = filter->shape[0];
        filter_c = filter->shape[1];
        filter_h = filter->shape[2];
        filter_w = filter->shape[3];
    }



    int out_n;
    int out_c;
    int out_h;
    int out_w;

    int pad_h = 0;
    int pad_w = 0;

    if (padding == 1) {
        pad_h = filter_h / 2;
        pad_w = filter_w / 2;
    }


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n, 
        input_c, 
        input_h, 
        input_w));


    //������
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        filter_n, 
        filter_c, 
        filter_h, 
        filter_w));


    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_descriptor,
        pad_h, pad_w, // zero-padding
        u, v, // stride
        1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //����output��4��ά��
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w));

   

    if (dataformat == 0) {
        assert(output->shape[0] == out_n);
        assert(output->shape[1] == out_c);
        assert(output->shape[2] == out_h);
        assert(output->shape[3] == out_w);
    }
    else {
        assert(output->shape[0] == out_n);
        assert(output->shape[3] == out_c);
        assert(output->shape[1] == out_h);
        assert(output->shape[2] == out_w);
    }


    //output��Ϣ
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        out_n,
        out_c,
        out_h,
        out_w));

    //�������㷨
    hipdnnConvolutionFwdAlgo_t algo;
    CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(handle,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        0,
        &algo));

    //׼����������Ŀռ�
    size_t workspace_size = 0;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(handle,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        algo,
        &workspace_size));
    void* workspace = nullptr;
    hipMalloc(&workspace, workspace_size);


    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionForward(handle,
        &alpha, //x*w����
        input_descriptor,
        input->data,
        filter_descriptor,
        filter->data,
        conv_descriptor,
        algo,
        workspace,
        workspace_size,
        &beta, //y����,y�������ݽ������ţ�
        output_descriptor,
        output->data));

    //�ڴ�
    hipFree(workspace);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;
}

int DLGpuConvolution2DForwardGetOutShape(const int* input_shapes,
    const int* filter_shapes,
    int* output_shapes,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int u,          /* vertical filter stride */
    const int v          /* horizontal filter stride */) {

   // cout << dataformat << endl;
   // cout << padding << endl;


    int input_n = input_shapes[0];
    int input_c = input_shapes[3];
    int input_h = input_shapes[1];
    int input_w = input_shapes[2];


    int filter_n = filter_shapes[0];
    int filter_c = filter_shapes[3];
    int filter_h = filter_shapes[1];
    int filter_w = filter_shapes[2];

    if (dataformat == 0) {
        input_n = input_shapes[0];
        input_c = input_shapes[1];
        input_h = input_shapes[2];
        input_w = input_shapes[3];

        filter_n = filter_shapes[0];
        filter_c = filter_shapes[1];
        filter_h = filter_shapes[2];
        filter_w = filter_shapes[3];
    }


    int out_n;
    int out_c;
    int out_h;
    int out_w;

    int pad_h = 0;
    int pad_w = 0;

    if (padding == 1) {
        pad_h = filter_h / 2;
        pad_w = filter_w / 2;
    }


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));


    //������
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        filter_n,
        filter_c,
        filter_h,
        filter_w));


    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_descriptor,
        pad_h, pad_w, // zero-padding
        u, v, // stride
        1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //����output��4��ά��
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w));



    if (dataformat == 0) {
        output_shapes[0] = out_n;
        output_shapes[1] = out_c;
        output_shapes[2] = out_h;
        output_shapes[3] = out_w;
    }
    else {
        output_shapes[0] = out_n;
        output_shapes[1] = out_h;
        output_shapes[2] = out_w;
        output_shapes[3] = out_c;
    }



    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));

    return 0;

}



int DLGpuConvolution2DBackward(const DLArrayHandle input,
    const DLArrayHandle doutput,
    const DLArrayHandle filter,
    DLArrayHandle dfilter,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int u,          /* vertical filter stride */
    const int v          /* horizontal filter stride */){

    assert(input->ndim == 4);
    assert(filter->ndim == 4);

    int input_n = input->shape[0];
    int input_c = input->shape[3];
    int input_h = input->shape[1];
    int input_w = input->shape[2];

    int filter_n = filter->shape[0];
    int filter_c = filter->shape[3];
    int filter_h = filter->shape[1];
    int filter_w = filter->shape[2];


    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = input->shape[2];
        input_w = input->shape[3];

        filter_n = filter->shape[0];
        filter_c = filter->shape[1];
        filter_h = filter->shape[2];
        filter_w = filter->shape[3];
    }





    int out_n;
    int out_c;
    int out_h;
    int out_w;

    int pad_h = 0;
    int pad_w = 0;

    if (padding == 1) {
        pad_h = filter_h / 2;
        pad_w = filter_w / 2;
    }


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));


    //������
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        filter_n,
        filter_c,
        filter_h,
        filter_w));


    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_descriptor,
        pad_h, pad_w, // zero-padding
        u, v, // stride
        1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //����output��4��ά��
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w));

    assert(doutput->shape[0] == out_n);
    assert(doutput->shape[1] == out_c);
    assert(doutput->shape[2] == out_h);
    assert(doutput->shape[3] == out_w);




    //output��Ϣ
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        out_n,
        out_c,
        out_h,
        out_w));

    hipdnnConvolutionBwdFilterAlgo_t  algo1;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterAlgorithm(handle,
        input_descriptor,
        output_descriptor,
        conv_descriptor,
        filter_descriptor,
        HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
        0,
        &algo1));

    hipdnnConvolutionBwdDataAlgo_t algo2;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataAlgorithm(handle,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
        0,
        &algo2));

    //׼����������Ŀռ�


    size_t workspace_size1= 0;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterWorkspaceSize(handle,
        input_descriptor,
        output_descriptor,
        conv_descriptor,
        filter_descriptor,
        algo1,
        &workspace_size1));
    void* workspace1= nullptr;
    hipMalloc(&workspace1, workspace_size1);

    size_t workspace_size2 = 0;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(handle,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        algo2,
        &workspace_size2));
    void* workspace2 = nullptr;
    hipMalloc(&workspace2, workspace_size2);



    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionBackwardFilter(handle,
        &alpha, //x*w����
        input_descriptor,
        input->data,
        output_descriptor,
        doutput->data,
        conv_descriptor,
        algo1,
        workspace1,
        workspace_size1,
        &beta, //y����,y�������ݽ������ţ�
        filter_descriptor,
        dfilter->data));



    CUDNN_CALL(hipdnnConvolutionBackwardData(handle,
        &alpha, //x*w����
        filter_descriptor,
        filter->data,
        output_descriptor,
        doutput->data,
        conv_descriptor,
        algo2,
        workspace2,
        workspace_size2,
        &beta, //y����,y�������ݽ������ţ�
        input_descriptor,
        dinput->data));


    //�ڴ�
    hipFree(workspace1);
    hipFree(workspace2);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;
}

//5ά
int DLGpuConvolution3DForward(const DLArrayHandle input,
    const DLArrayHandle filter,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int s1,          
    const int s2,     
    const int s3) {

    assert(input->ndim == 5);
    assert(filter->ndim == 5);

    int* input_shape, * output_shape, * filter_shape, * inputstrides,*outputstrides;

    int* padA, * filterStrideA, * dilationA;

    input_shape = (int*)malloc(sizeof(int) * 5);
    filter_shape = (int*)malloc(sizeof(int) * 5);
    output_shape = (int*)malloc(sizeof(int) * 5);
    inputstrides = (int*)malloc(sizeof(int) * 5);
    outputstrides = (int*)malloc(sizeof(int) * 5);
    padA = (int*)malloc(sizeof(int) * 3);
    filterStrideA = (int*)malloc(sizeof(int) * 3);
    dilationA = (int*)malloc(sizeof(int) * 3);

    for (int i=0;i<5;i++)
    {
        input_shape[i]=input->shape[i];
        filter_shape[i]=filter->shape[i];
    }


    for (int i = 0; i < 3; i++) {
        padA[i] = 0;
        dilationA[i] = 1;
    }

    if (padding == 1) {
        for (int i = 0; i < 3; i++) {
            padA[i] = filter_shape[i+2]/2;
            
        }
    }


    filterStrideA[0] = s1;
    filterStrideA[1] = s2;
    filterStrideA[2] = s3;


    inputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        inputstrides[4 - i] = inputstrides[5 - i] * input_shape[5 - i];
    }



     hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        input_shape));

    //�˺���
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilterNdDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        5,
        filter_shape));

    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolutionNdDescriptor(conv_descriptor,
        3,
        padA,
        filterStrideA,
        dilationA,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //output��shape
    CUDNN_CALL(cudnnGetConvolutionNdForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        5,
        output_shape));


    assert(output->shape[0] == output_shape[0]);
    assert(output->shape[1] == output_shape[1]);
    assert(output->shape[2] == output_shape[2]);
    assert(output->shape[3] == output_shape[3]);
    assert(output->shape[4] == output_shape[4]);


    outputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        outputstrides[4 - i] = outputstrides[5 - i] * output_shape[5 - i];
    }


    //output
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        output_shape));

    //�������㷨
    hipdnnConvolutionFwdAlgo_t algo;
    CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(handle,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        0,
        &algo));

    //׼����������Ŀռ�
    size_t workspace_size = 0;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(handle,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        algo,
        &workspace_size));
    void* workspace = nullptr;
    hipMalloc(&workspace, workspace_size);


    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionForward(handle,
        &alpha, //x*w����
        input_descriptor,
        input->data,
        filter_descriptor,
        filter->data,
        conv_descriptor,
        algo,
        workspace,
        workspace_size,
        &beta, //y����,y�������ݽ������ţ�
        output_descriptor,
        output->data));

    //�ڴ�
    hipFree(workspace);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));


   return 0;




}


int DLGpuConvolution3DForwardGetOutShape(const int* input_shapes,
    const int* filter_shapes,
    int* output_shapes,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int s1,
    const int s2,
    const int s3) {

    int* padA, * filterStrideA, * dilationA;

    padA = (int*)malloc(sizeof(int) * 3);
    filterStrideA = (int*)malloc(sizeof(int) * 3);
    dilationA = (int*)malloc(sizeof(int) * 3);


    for (int i = 0; i < 3; i++) {
        padA[i] = 0;
        dilationA[i] = 1;

    }

    if (padding == 1) {
        for (int i = 0; i < 3; i++) {
            padA[i] = filter_shapes[i+2]/2;

        }
    }

    filterStrideA[0] = s1;
    filterStrideA[1] = s2;
    filterStrideA[2] = s3;


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        input_shapes));

    //�˺���
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilterNdDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        5,
        filter_shapes));


    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolutionNdDescriptor(conv_descriptor,
        3,
        padA,
        filterStrideA,
        dilationA,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //output��shape
    CUDNN_CALL(cudnnGetConvolutionNdForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        5,
        output_shapes));



    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));

    return 0;

}

int DLGpuConvolution3DBackward(const DLArrayHandle input,
    const DLArrayHandle doutput,
    const DLArrayHandle filter,
    DLArrayHandle dfilter,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int s1,
    const int s2,
    const int s3) {

    assert(input->ndim == 5);
    assert(filter->ndim == 5);

    int* input_shape, * output_shape, * filter_shape, * inputstrides,*outputstrides;

    int* padA, * filterStrideA, * dilationA;

    input_shape = (int*)malloc(sizeof(int) * 5);
    filter_shape = (int*)malloc(sizeof(int) * 5);
    output_shape = (int*)malloc(sizeof(int) * 5);
    inputstrides = (int*)malloc(sizeof(int) * 5);
    outputstrides = (int*)malloc(sizeof(int) * 5);
    padA = (int*)malloc(sizeof(int) * 3);
    filterStrideA = (int*)malloc(sizeof(int) * 3);
    dilationA = (int*)malloc(sizeof(int) * 3);

    for (int i=0;i<5;i++)
    {
        input_shape[i]=input->shape[i];
        filter_shape[i]=filter->shape[i];
    }


    for (int i = 0; i < 3; i++) {
        padA[i] = 0;
        dilationA[i] = 1;
    }

    if (padding == 1) {
        for (int i = 0; i < 3; i++) {
            padA[i] = filter_shape[i+2]/2;

        }
    }


    filterStrideA[0] = s1;
    filterStrideA[1] = s2;
    filterStrideA[2] = s3;


    inputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        inputstrides[4 - i] = inputstrides[5 - i] * input_shape[5 - i];
    }



     hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        input_shape));

    //�˺���
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilterNdDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        5,
        filter_shape));

    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolutionNdDescriptor(conv_descriptor,
        3,
        padA,
        filterStrideA,
        dilationA,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //output��shape
    CUDNN_CALL(cudnnGetConvolutionNdForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        5,
        output_shape));


    assert(doutput->shape[0] == output_shape[0]);
    assert(doutput->shape[1] == output_shape[1]);
    assert(doutput->shape[2] == output_shape[2]);
    assert(doutput->shape[3] == output_shape[3]);
    assert(doutput->shape[4] == output_shape[4]);


    outputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        outputstrides[4 - i] = outputstrides[5 - i] * output_shape[5 - i];
    }


    //output
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        output_shape));

       hipdnnConvolutionBwdFilterAlgo_t  algo1;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterAlgorithm(handle,
        input_descriptor,
        output_descriptor,
        conv_descriptor,
        filter_descriptor,
        HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
        0,
        &algo1));

    hipdnnConvolutionBwdDataAlgo_t algo2;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataAlgorithm(handle,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
        0,
        &algo2));

    //׼����������Ŀռ�


    size_t workspace_size1= 0;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterWorkspaceSize(handle,
        input_descriptor,
        output_descriptor,
        conv_descriptor,
        filter_descriptor,
        algo1,
        &workspace_size1));
    void* workspace1= nullptr;
    hipMalloc(&workspace1, workspace_size1);

    size_t workspace_size2 = 0;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(handle,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        algo2,
        &workspace_size2));
    void* workspace2 = nullptr;
    hipMalloc(&workspace2, workspace_size2);



    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionBackwardFilter(handle,
        &alpha, //x*w����
        input_descriptor,
        input->data,
        output_descriptor,
        doutput->data,
        conv_descriptor,
        algo1,
        workspace1,
        workspace_size1,
        &beta, //y����,y�������ݽ������ţ�
        filter_descriptor,
        dfilter->data));



    CUDNN_CALL(hipdnnConvolutionBackwardData(handle,
        &alpha, //x*w����
        filter_descriptor,
        filter->data,
        output_descriptor,
        doutput->data,
        conv_descriptor,
        algo2,
        workspace2,
        workspace_size2,
        &beta, //y����,y�������ݽ������ţ�
        input_descriptor,
        dinput->data));


    //�ڴ�
    hipFree(workspace1);
    hipFree(workspace2);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;
}



int DLGpuPooling1DForward(const DLArrayHandle input,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    hipdnnPoolingMode_t poolingMode,
    const int padding_w,
    const int v,
    const int filter_w) {

    int padding_h = 0;
    int u = 1;
    int filter_h = 1;


    int input_n = input->shape[0];
    int input_c = input->shape[2];
    int input_h = 1;
    int input_w = input->shape[1];

   



    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = 1;
        input_w = input->shape[2];
    }




    int output_n;
    int output_c;
    int output_h;
    int output_w;


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_descriptor,
        poolingMode,
        HIPDNN_NOT_PROPAGATE_NAN,
        filter_h, filter_w,
        padding_h, padding_w, // zero-padding
        u, v // stride
    ));


    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));



    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        &output_n,
        &output_c,
        &output_h,
        &output_w));

    if (dataformat == 0) {
        assert(output->shape[0] == output_n);
        assert(output->shape[1] == output_c);
        assert(1 == output_h);
        assert(output->shape[2] == output_w);
    }
    else {
        assert(output->shape[0] == output_n);
        assert(output->shape[2] == output_c);
        assert(1 == output_h);
        assert(output->shape[1] == output_w);
    }
    

    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        output_n,
        output_c,
        output_h,
        output_w));


    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnPoolingForward(handle,
        pool_descriptor,
        &alpha,
        input_descriptor,
        input->data,
        &beta,
        output_descriptor,
        output->data));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;
}


int DLGpuPooling1DForwardGetOutShape(const int* input_shapes,
    int* output_shapes,
    hipdnnTensorFormat_t dataformat,
    hipdnnPoolingMode_t poolingMode,
    const int padding_w,
    const int v,
    const int filter_w){



    int padding_h = 0;
    int u = 1;
    int filter_h = 1;

    int input_n = input_shapes[0];
    int input_c = input_shapes[2];
    int input_h = 1;
    int input_w = input_shapes[1];



    if (dataformat == 0) {
        input_n = input_shapes[0];
        input_c = input_shapes[1];
        input_h = 1;
        input_w = input_shapes[2];
    }

    int output_n;
    int output_c;
    int output_h;
    int output_w;



    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_descriptor,
        poolingMode,
        HIPDNN_NOT_PROPAGATE_NAN,
        filter_h, filter_w,
        padding_h, padding_w, // zero-padding
        u, v // stride
    ));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));


    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        &output_n,
        &output_c,
        &output_h,
        &output_w));


    if (dataformat == 0) {
        output_shapes[0] = output_n;
        output_shapes[1] = output_c;
        output_shapes[2] = output_w;
    }
    else {
        output_shapes[0] = output_n;
        output_shapes[1] = output_w;
        output_shapes[2] = output_c;
    }

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;


}

int DLGpuPooling1DBackward(const DLArrayHandle input,
    const DLArrayHandle output,
    const DLArrayHandle doutput,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    hipdnnPoolingMode_t poolingMode,
    const int padding_w,
    const int v,
    const int filter_w) {

    int padding_h = 0;
    int u = 1;
    int filter_h = 1;


    int input_n = input->shape[0];
    int input_c = input->shape[2];
    int input_h = 1;
    int input_w = input->shape[1];

    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = 1;
        input_w = input->shape[2];
    }


    int output_n;
    int output_c;
    int output_h;
    int output_w;


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_descriptor,
        poolingMode,
        HIPDNN_NOT_PROPAGATE_NAN,
        filter_h, filter_w,
        padding_h, padding_w, // zero-padding
        u, v // stride
    ));


    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));



    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        &output_n,
        &output_c,
        &output_h,
        &output_w));


   

    if (dataformat == 0) {
        assert(output->shape[0] == output_n);
        assert(output->shape[1] == output_c);
        assert(1 == output_h);
        assert(output->shape[2] == output_w);
    }
    else {
        assert(output->shape[0] == output_n);
        assert(output->shape[2] == output_c);
        assert(1 == output_h);
        assert(output->shape[1] == output_w);
    }

    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        output_n,
        output_c,
        output_h,
        output_w));


    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnPoolingBackward(handle,
        pool_descriptor,
        &alpha,
        output_descriptor,
        output->data,
        output_descriptor,
        doutput->data,
        input_descriptor,
        input->data,
        &beta,
        input_descriptor,
        dinput->data));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;
}



int DLGpuPooling2DForward(const DLArrayHandle input,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    hipdnnPoolingMode_t poolingMode,
    const int padding_h,
    const int padding_w,
    const int u,
    const int v,
    const int filter_h,
    const int filter_w)
{

    int input_n = input->shape[0];
    int input_c = input->shape[3];
    int input_h = input->shape[1];
    int input_w = input->shape[2];
    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = input->shape[2];
        input_w = input->shape[3];
    }



    int output_n;
    int output_c;
    int output_h;
    int output_w;

    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_descriptor,
        poolingMode,
        HIPDNN_NOT_PROPAGATE_NAN,
        filter_h, filter_w,
        padding_h, padding_w, // zero-padding
        u, v // stride
    ));
    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));




    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        &output_n,
        &output_c,
        &output_h,
        &output_w));

    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        output_n,
        output_c,
        output_h,
        output_w));


    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnPoolingForward(handle,
        pool_descriptor,
        &alpha,
        input_descriptor,
        input->data,
        &beta,
        output_descriptor,
        output->data));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
   return 0;
}

int DLGpuPooling2DForwardGetOutShape(const int* input_shapes,
    int* output_shapes,
    hipdnnTensorFormat_t dataformat,
    hipdnnPoolingMode_t poolingMode,
    const int padding_h,
    const int padding_w,
    const int u,
    const int v,
    const int filter_h,
    const int filter_w){



    int input_n = input_shapes[0];
    int input_c = input_shapes[3];
    int input_h = input_shapes[1];
    int input_w = input_shapes[2];
    if (dataformat == 0) {
        input_n = input_shapes[0];
        input_c = input_shapes[1];
        input_h = input_shapes[2];
        input_w = input_shapes[3];
    }



    int output_n;
    int output_c;
    int output_h;
    int output_w;



    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_descriptor,
        poolingMode,
        HIPDNN_NOT_PROPAGATE_NAN,
        filter_h, filter_w,
        padding_h, padding_w, // zero-padding
        u, v // stride
    ));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));


    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        &output_n,
        &output_c,
        &output_h,
        &output_w));


    if (dataformat == 0) {
        output_shapes[0] = output_n;
        output_shapes[1] = output_c;
        output_shapes[2] = output_h;
        output_shapes[3] = output_w;
    }
    else {
        output_shapes[0] = output_n;
        output_shapes[1] = output_h;
        output_shapes[2] = output_w;
        output_shapes[3] = output_c;
    }

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;


}

int DLGpuPooling2DBackward(const DLArrayHandle input,
    const DLArrayHandle output,
    const DLArrayHandle doutput,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    hipdnnPoolingMode_t poolingMode,
    const int padding_h,
    const int padding_w,
    const int u,
    const int v,
    const int filter_h,
    const int filter_w)
{
    int input_n = input->shape[0];
    int input_c = input->shape[3];
    int input_h = input->shape[1];
    int input_w = input->shape[2];

    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = input->shape[2];
        input_w = input->shape[3];
    }



    int output_n;
    int output_c;
    int output_h;
    int output_w;

    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_descriptor,
        poolingMode,
        HIPDNN_NOT_PROPAGATE_NAN,
        filter_h, filter_w,
        padding_h, padding_w, // zero-padding
        u, v // stride
    ));
    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));




    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        &output_n,
        &output_c,
        &output_h,
        &output_w));

    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        output_n,
        output_c,
        output_h,
        output_w));


    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnPoolingBackward(handle,
        pool_descriptor,
        &alpha,
        output_descriptor,
        output->data,
        output_descriptor,
        doutput->data,
        input_descriptor,
        input->data,
        &beta,
        input_descriptor,
        dinput->data));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
   return 0;
}




int DLGpuPooling3DForward(const DLArrayHandle input,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    hipdnnPoolingMode_t poolingMode,
    const int padding1,
    const int padding2,
    const int padding3,
    const int s1,
    const int s2,
    const int s3,
    const int filter1,
    const int filter2,
    const int filter3)
{
    assert(input->ndim == 5);


    int* input_shape, * output_shape, * filter_shape, * inputstrides, * outputstrides;


    int* padA, * filterStrideA;
    input_shape = (int*)malloc(sizeof(int) * 5);
    filter_shape = (int*)malloc(sizeof(int) * 3);
    output_shape = (int*)malloc(sizeof(int) * 5);
    inputstrides = (int*)malloc(sizeof(int) * 5);
    outputstrides = (int*)malloc(sizeof(int) * 5);
    padA = (int*)malloc(sizeof(int) * 3);
    filterStrideA = (int*)malloc(sizeof(int) * 3);
    for(int i=0;i<5;i++)
    {
        input_shape[i]= input->shape[i];
    }
    filter_shape[0] = filter1;
    filter_shape[1] = filter2;
    filter_shape[2] = filter3;
    filterStrideA[0] = s1;
    filterStrideA[1] = s2;
    filterStrideA[2] = s3;
    padA[0] = padding1;
    padA[1] = padding2;
    padA[2] = padding3;


    inputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        inputstrides[4 - i] = inputstrides[5 - i] * input_shape[5 - i];
    }

    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPoolingNdDescriptor(pool_descriptor,
        poolingMode,
        HIPDNN_NOT_PROPAGATE_NAN,
        3,
        filter_shape,
        padA,
        filterStrideA));
    

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        input_shape));


    CUDNN_CALL(cudnnGetPoolingNdForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        5,
        output_shape));

    assert(output->shape[0] == output_shape[0]);
    assert(output->shape[1] == output_shape[1]);
    assert(output->shape[2] == output_shape[2]);
    assert(output->shape[3] == output_shape[3]);
    assert(output->shape[4] == output_shape[4]);

    outputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        outputstrides[4 - i] = outputstrides[5 - i] * output_shape[5 - i];
    }


    //output
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        output_shape));


    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnPoolingForward(handle,
        pool_descriptor,
        &alpha,
        input_descriptor,
        input->data,
        &beta,
        output_descriptor,
        output->data));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;

}

int DLGpuPooling3DForwardGetOutShape(const int* input_shapes,
    int* output_shapes,
    hipdnnTensorFormat_t dataformat,
    hipdnnPoolingMode_t poolingMode,
    const int padding1,
    const int padding2,
    const int padding3,
    const int s1,
    const int s2,
    const int s3,
    const int filter1,
    const int filter2,
    const int filter3){

    int* filter_shape,*padA,*filterStrideA;

    filter_shape = (int*)malloc(sizeof(int) * 3);
    padA = (int*)malloc(sizeof(int) * 3);
    filterStrideA = (int*)malloc(sizeof(int) * 3);
    filter_shape[0] = filter1;
    filter_shape[1] = filter2;
    filter_shape[2] = filter3;
    filterStrideA[0] = s1;
    filterStrideA[1] = s2;
    filterStrideA[2] = s3;
    padA[0] = padding1;
    padA[1] = padding2;
    padA[2] = padding3;



    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


     hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPoolingNdDescriptor(pool_descriptor,
        poolingMode,
        HIPDNN_NOT_PROPAGATE_NAN,
        3,
        filter_shape,
        padA,
        filterStrideA));


    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        input_shapes));


    CUDNN_CALL(cudnnGetPoolingNdForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        5,
        output_shapes));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;


}


int DLGpuPooling3DBackward(const DLArrayHandle input,
    const DLArrayHandle output,
    const DLArrayHandle doutput,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    hipdnnPoolingMode_t poolingMode,
    const int padding1,
    const int padding2,
    const int padding3,
    const int s1,
    const int s2,
    const int s3,
    const int filter1,
    const int filter2,
    const int filter3)
{
    assert(input->ndim == 5);


    int* input_shape, * output_shape, * filter_shape, * inputstrides, * outputstrides;


    int* padA, * filterStrideA;
    input_shape = (int*)malloc(sizeof(int) * 5);
    filter_shape = (int*)malloc(sizeof(int) * 3);
    output_shape = (int*)malloc(sizeof(int) * 5);
    inputstrides = (int*)malloc(sizeof(int) * 5);
    outputstrides = (int*)malloc(sizeof(int) * 5);
    padA = (int*)malloc(sizeof(int) * 3);
    filterStrideA = (int*)malloc(sizeof(int) * 3);
    for(int i=0;i<5;i++)
    {
        input_shape[i]= input->shape[i];
    }
    filter_shape[0] = filter1;
    filter_shape[1] = filter2;
    filter_shape[2] = filter3;
    filterStrideA[0] = s1;
    filterStrideA[1] = s2;
    filterStrideA[2] = s3;
    padA[0] = padding1;
    padA[1] = padding2;
    padA[2] = padding3;


    inputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        inputstrides[4 - i] = inputstrides[5 - i] * input_shape[5 - i];
    }

    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPoolingNdDescriptor(pool_descriptor,
        poolingMode,
        HIPDNN_NOT_PROPAGATE_NAN,
        3,
        filter_shape,
        padA,
        filterStrideA));


    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        input_shape));


    CUDNN_CALL(cudnnGetPoolingNdForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        5,
        output_shape));

    assert(output->shape[0] == output_shape[0]);
    assert(output->shape[1] == output_shape[1]);
    assert(output->shape[2] == output_shape[2]);
    assert(output->shape[3] == output_shape[3]);
    assert(output->shape[4] == output_shape[4]);

    outputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        outputstrides[4 - i] = outputstrides[5 - i] * output_shape[5 - i];
    }


    //output
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        output_shape));


    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnPoolingBackward(handle,
        pool_descriptor,
        &alpha,
        output_descriptor,
        output->data,
        output_descriptor,
        doutput->data,
        input_descriptor,
        input->data,
        &beta,
        input_descriptor,
        dinput->data));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;

}






//activation
int DLGpuActivationForward(const DLArrayHandle input,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    hipdnnActivationMode_t activationMode) {

    assert(input->ndim==4||input->ndim==3||input->ndim==5);

    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));
    //input
    hipdnnTensorDescriptor_t input_descriptor;

    int input_n;
    int input_c;
    int input_h;
    int input_w;

    if(input->ndim == 3){
        if (dataformat == 0) {
            input_n = input->shape[0];
            input_c = input->shape[1];
            input_h = 1;
            input_w = input->shape[2];
        }else{
            input_n = input->shape[0];
            input_c = input->shape[2];
            input_h = 1;
            input_w = input->shape[1];
        }
        //input
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
            dataformat,
            HIPDNN_DATA_FLOAT,
            input_n,
            input_c,
            input_h,
            input_w));

    }


    if(input->ndim == 4){
        if (dataformat == 0) {
            input_n = input->shape[0];
            input_c = input->shape[1];
            input_h = input->shape[2];
            input_w = input->shape[3];
        }else{
            input_n = input->shape[0];
            input_c = input->shape[3];
            input_h = input->shape[1];
            input_w = input->shape[2];
        }
        //input
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
            dataformat,
            HIPDNN_DATA_FLOAT,
            input_n,
            input_c,
            input_h,
            input_w));

    }

    if(input->ndim == 5){

        int* input_shape;
        input_shape = (int*)malloc(sizeof(int) * 5);
        for(int i=0;i<5;i++){
            input_shape[i]= input->shape[i];
        }

        //input
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
        CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
            dataformat,
            HIPDNN_DATA_FLOAT,
            5,
            input_shape));

        

    }
    auto alpha = 1.0f, beta = 0.0f;
    if (activationMode != 3 ){
        // 描述激活
        hipdnnActivationDescriptor_t activation_descriptor;
        CUDNN_CALL(hipdnnCreateActivationDescriptor(&activation_descriptor));
        CUDNN_CALL(hipdnnSetActivationDescriptor(activation_descriptor,
            activationMode,
            HIPDNN_PROPAGATE_NAN,
            /*relu_coef=*/0));

       
        CUDNN_CALL(hipdnnActivationForward(handle,
            activation_descriptor,
            &alpha,
            input_descriptor,
            input->data,
            &beta,
            input_descriptor,
            output->data));
        CUDNN_CALL(hipdnnDestroyActivationDescriptor(activation_descriptor));
       
    }else{

        CUDNN_CALL(hipdnnSoftmaxForward(handle,
            HIPDNN_SOFTMAX_FAST,
            HIPDNN_SOFTMAX_MODE_INSTANCE,
            &alpha,
            input_descriptor,
            input->data,
            &beta,
            input_descriptor,
            output->data));
    }


    
   
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;



}


//activation
int DLGpuActivationBackward(const DLArrayHandle input,
    DLArrayHandle dinput,
    const DLArrayHandle output,
    const DLArrayHandle doutput,
    hipdnnTensorFormat_t dataformat,
    hipdnnActivationMode_t activationMode) {

    assert(input->ndim==4||input->ndim==3||input->ndim==5);

    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));
    //input
    hipdnnTensorDescriptor_t input_descriptor;

    int input_n;
    int input_c;
    int input_h;
    int input_w;

    if(input->ndim == 3){
        if (dataformat == 0) {
            input_n = input->shape[0];
            input_c = input->shape[1];
            input_h = 1;
            input_w = input->shape[2];
        }else{
            input_n = input->shape[0];
            input_c = input->shape[2];
            input_h = 1;
            input_w = input->shape[1];
        }
        //input
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
            dataformat,
            HIPDNN_DATA_FLOAT,
            input_n,
            input_c,
            input_h,
            input_w));

    }


    if(input->ndim == 4){
        if (dataformat == 0) {
            input_n = input->shape[0];
            input_c = input->shape[1];
            input_h = input->shape[2];
            input_w = input->shape[3];
        }else{
            input_n = input->shape[0];
            input_c = input->shape[3];
            input_h = input->shape[1];
            input_w = input->shape[2];
        }
        //input
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
            dataformat,
            HIPDNN_DATA_FLOAT,
            input_n,
            input_c,
            input_h,
            input_w));

    }

    if(input->ndim == 5){

        int* input_shape;
        input_shape = (int*)malloc(sizeof(int) * 5);
        for(int i=0;i<5;i++){
            input_shape[i]= input->shape[i];
        }

        //input
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
        CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
            dataformat,
            HIPDNN_DATA_FLOAT,
            5,
            input_shape));

        

    }
    auto alpha = 1.0f, beta = 0.0f;
    if(activationMode != 3){

        // 描述激活
        hipdnnActivationDescriptor_t activation_descriptor;
        CUDNN_CALL(hipdnnCreateActivationDescriptor(&activation_descriptor));
        CUDNN_CALL(hipdnnSetActivationDescriptor(activation_descriptor,
            activationMode,
            HIPDNN_PROPAGATE_NAN,
            /*relu_coef=*/0));

        // 激活函数求导
       
        CUDNN_CALL(hipdnnActivationBackward(handle,
            activation_descriptor,
            &alpha,
            input_descriptor,
            output->data,
            input_descriptor,
            doutput->data,
            input_descriptor,
            input->data,
            &beta,
            input_descriptor,
            dinput->data));
        CUDNN_CALL(hipdnnDestroyActivationDescriptor(activation_descriptor));
    }else{


        CUDNN_CALL(hipdnnSoftmaxBackward(handle,
            HIPDNN_SOFTMAX_FAST,
            HIPDNN_SOFTMAX_MODE_INSTANCE,
            &alpha,
            input_descriptor,
            output->data,
            input_descriptor,
            doutput->data,
            &beta,
            input_descriptor,
            dinput->data));


    }

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;



}

//dropout
int DLGpuDropoutForward(const DLArrayHandle input,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    const float dropout,
    const int seed,
    void **reserveSpace_p/*back use*/){



    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;

    int input_n;
    int input_c;
    int input_h;
    int input_w;

    if(input->ndim == 3){
        if (dataformat == 0) {
            input_n = input->shape[0];
            input_c = input->shape[1];
            input_h = 1;
            input_w = input->shape[2];
        }else{
            input_n = input->shape[0];
            input_c = input->shape[2];
            input_h = 1;
            input_w = input->shape[1];
        }
        //input
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
            dataformat,
            HIPDNN_DATA_FLOAT,
            input_n,
            input_c,
            input_h,
            input_w));

    }


    if(input->ndim == 4){
        if (dataformat == 0) {
            input_n = input->shape[0];
            input_c = input->shape[1];
            input_h = input->shape[2];
            input_w = input->shape[3];
        }else{
            input_n = input->shape[0];
            input_c = input->shape[3];
            input_h = input->shape[1];
            input_w = input->shape[2];
        }
        //input
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
            dataformat,
            HIPDNN_DATA_FLOAT,
            input_n,
            input_c,
            input_h,
            input_w));

    }

    if(input->ndim == 5){

        int* input_shape;
        input_shape = (int*)malloc(sizeof(int) * 5);
        for(int i=0;i<5;i++){
            input_shape[i]= input->shape[i];
        }

        //input
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
        CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
            dataformat,
            HIPDNN_DATA_FLOAT,
            5,
            input_shape));

        

    }
    size_t stateSizeInBytes = 1;
    size_t reserveSpaceSizeInBytes = 1;
    void *states;
    //unsigned long long  seed = 0;//ini rand seed
    CUDNN_CALL(hipdnnDropoutGetStatesSize(handle,
        &stateSizeInBytes));

    CUDNN_CALL(cudnnDropoutGetReserveSpaceSize(input_descriptor,
        &reserveSpaceSizeInBytes));


    hipMalloc((void**)&states, stateSizeInBytes);
    hipMalloc((void**)reserveSpace_p, reserveSpaceSizeInBytes);




    hipdnnDropoutDescriptor_t dropout_descriptor;
    CUDNN_CALL(hipdnnCreateDropoutDescriptor(&dropout_descriptor));

    CUDNN_CALL(hipdnnSetDropoutDescriptor(dropout_descriptor,
        handle,
        dropout,
        states,
        stateSizeInBytes,
        seed));


    CUDNN_CALL(cudnnDropoutForward(handle,
        dropout_descriptor,
        input_descriptor,
        input->data,
        input_descriptor,
        output->data,
        *reserveSpace_p,
        reserveSpaceSizeInBytes));

    CUDNN_CALL(hipdnnDestroyDropoutDescriptor(dropout_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    hipFree(states);
    return 0;

}




int DLGpuDropoutBackward(const DLArrayHandle doutput,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    const float dropout,
    const int seed,
    void **reserveSpace_p/*back use*/){






    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;

    int input_n;
    int input_c;
    int input_h;
    int input_w;

    if(dinput->ndim == 3){
        if (dataformat == 0) {
            input_n = dinput->shape[0];
            input_c = dinput->shape[1];
            input_h = 1;
            input_w = dinput->shape[2];
        }else{
            input_n = dinput->shape[0];
            input_c = dinput->shape[2];
            input_h = 1;
            input_w = dinput->shape[1];
        }
        //input
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
            dataformat,
            HIPDNN_DATA_FLOAT,
            input_n,
            input_c,
            input_h,
            input_w));

    }


    if(dinput->ndim == 4){
        if (dataformat == 0) {
            input_n = dinput->shape[0];
            input_c = dinput->shape[1];
            input_h = dinput->shape[2];
            input_w = dinput->shape[3];
        }else{
            input_n = dinput->shape[0];
            input_c = dinput->shape[3];
            input_h = dinput->shape[1];
            input_w = dinput->shape[2];
        }
        //input
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
            dataformat,
            HIPDNN_DATA_FLOAT,
            input_n,
            input_c,
            input_h,
            input_w));

    }

    if(dinput->ndim == 5){

        int* input_shape;
        input_shape = (int*)malloc(sizeof(int) * 5);
        for(int i=0;i<5;i++){
            input_shape[i]= dinput->shape[i];
        }

        //input
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
        CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
            dataformat,
            HIPDNN_DATA_FLOAT,
            5,
            input_shape));

        

    }



    size_t stateSizeInBytes = 1;
    size_t reserveSpaceSizeInBytes = 1;
    void *states;
    //unsigned long long  seed = 0;//ini rand seed
    CUDNN_CALL(hipdnnDropoutGetStatesSize(handle,
        &stateSizeInBytes));

    CUDNN_CALL(cudnnDropoutGetReserveSpaceSize(input_descriptor,
        &reserveSpaceSizeInBytes));

    hipdnnDropoutDescriptor_t dropout_descriptor;
    CUDNN_CALL(hipdnnCreateDropoutDescriptor(&dropout_descriptor));

    hipMalloc((void**)&states, stateSizeInBytes);


    CUDNN_CALL(hipdnnSetDropoutDescriptor(dropout_descriptor,
        handle,
        dropout,
        states,
        stateSizeInBytes,
        seed));


    CUDNN_CALL(cudnnDropoutBackward(handle,
        dropout_descriptor,
        input_descriptor,
        doutput->data,
        input_descriptor,
        dinput->data,
        *reserveSpace_p,
        reserveSpaceSizeInBytes));


    CUDNN_CALL(hipdnnDestroyDropoutDescriptor(dropout_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    hipFree(states);
    hipFree(*reserveSpace_p);
    return 0;

}




//loss

__global__ void matrix_cross_entropy_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  input_b += y * ncol;
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
   output[x+y*ncol] = input_b[x] * log(input_a[x])+(1-input_b[x])* log(1-input_a[x]);
  }
  loss_per_row[y] = loss;
  /*__syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }

    output[0] = mean_loss;
  }*/
}
__global__ void matrix_l1loss_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  input_b += y * ncol;
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
    loss += abs(input_b[x]-input_a[x]);
  }
  loss_per_row[y] = loss;
  __syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }
    mean_loss /= nrow;
    output[0] = mean_loss;
  }
}
__global__ void matrix_l2loss_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  input_b += y * ncol;
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
    loss += pow(input_b[x]-input_a[x],2);
  }
  loss_per_row[y] = loss;
  __syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }
    mean_loss /= nrow;
    mean_loss/=2;
    output[0] = mean_loss;
  }
}

__global__ void matrix_l1lossgradient_kernel(const float* inputArr,const float* inputArr1, const float* gradArr,
                                            float* outputArr, int count,int n) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = (inputArr[index]-inputArr1[index])> 0 ? gradArr[0]/n : -gradArr[0] /n;
    }
}
__global__ void matrix_l2lossgradient_kernel(const float* inputArr,const float* inputArr1, const float* gradArr,
                                            float* outputArr, int count,int n) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = (inputArr[index]-inputArr1[index])*gradArr[0]/n;
    }
}
__global__ void matrix_l1regular_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
    loss += abs(input_a[x]);
  }
  loss_per_row[y] = loss;
  __syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }
    mean_loss /= nrow;
    output[0] = mean_loss;
  }
}
__global__ void matrix_l2regular_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
    loss += pow(input_a[x],2);
  }
  loss_per_row[y] = loss;
  __syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }
    mean_loss /= nrow;
    mean_loss/=2;
    output[0] = mean_loss;
  }
}
__global__ void matrix_l1regulargradient_kernel(const float* inputArr, const float* gradArr,
                                            float* outputArr, int count,int n) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = inputArr[index]> 0 ? gradArr[0]/n : -gradArr[0]/n;
    }
}
__global__ void matrix_l2regulargradient_kernel(const float* inputArr, const float* gradArr,
                                            float* outputArr, int count,int n) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = inputArr[index]*gradArr[0]/n;
    }
}


int DLGpuCrossEntropy(const DLArrayHandle input_a,
    const DLArrayHandle input_b,
    DLArrayHandle output) {
    assert(input_a->ndim == 2);
    assert(input_b->ndim == 2);
    assert(output->ndim == 2);
    assert(input_a->shape[0] == input_b->shape[0] &&
        input_a->shape[1] == input_b->shape[1]);
    int nrow = input_a->shape[0];
    assert(nrow <= 1024 * 4);
    int ncol = input_a->shape[1];
    const float* input_data_a = (const float*)input_a->data;
    const float* input_data_b = (const float*)input_b->data;
    float* output_data = (float*)output->data;
    dim3 threads;
    if (nrow <= 1024) {
        threads.x = nrow;
    }
    else {
        threads.x = 1024;
        threads.y = (nrow + 1023) / 1024;
    }
    // 1 block, each block with 'threads' number of threads with 'nrow' shared
    // memory size
    matrix_cross_entropy_kernel << <1, threads, nrow * sizeof(float) >> > (
        nrow, ncol, input_data_a, input_data_b, output_data);
    return 0;
}
int DLGpuL1loss(const DLArrayHandle input_a,
    const DLArrayHandle input_b,
    DLArrayHandle output) {
    assert(input_a->ndim == 2);
    assert(input_b->ndim == 2);
    assert(output->ndim == 1);
    assert(input_a->shape[0] == input_b->shape[0] &&
        input_a->shape[1] == input_b->shape[1]);
    int nrow = input_a->shape[0];
    assert(nrow <= 1024 * 4);
    int ncol = input_a->shape[1];
    const float* input_data_a = (const float*)input_a->data;
    const float* input_data_b = (const float*)input_b->data;
    float* output_data = (float*)output->data;
    dim3 threads;
    if (nrow <= 1024) {
        threads.x = nrow;
    }
    else {
        threads.x = 1024;
        threads.y = (nrow + 1023) / 1024;
    }
    // 1 block, each block with 'threads' number of threads with 'nrow' shared
    // memory size
    matrix_l1loss_kernel << <1, threads, nrow * sizeof(float) >> > (
        nrow, ncol, input_data_a, input_data_b, output_data);
    return 0;
}
int DLGpuL2loss(const DLArrayHandle input_a,
    const DLArrayHandle input_b,
    DLArrayHandle output) {
    assert(input_a->ndim == 2);
    assert(input_b->ndim == 2);
    assert(output->ndim == 1);
    assert(input_a->shape[0] == input_b->shape[0] &&
        input_a->shape[1] == input_b->shape[1]);
    int nrow = input_a->shape[0];
    assert(nrow <= 1024 * 4);
    int ncol = input_a->shape[1];
    const float* input_data_a = (const float*)input_a->data;
    const float* input_data_b = (const float*)input_b->data;
    float* output_data = (float*)output->data;
    dim3 threads;
    if (nrow <= 1024) {
        threads.x = nrow;
    }
    else {
        threads.x = 1024;
        threads.y = (nrow + 1023) / 1024;
    }
    // 1 block, each block with 'threads' number of threads with 'nrow' shared
    // memory size
    matrix_l2loss_kernel << <1, threads, nrow * sizeof(float) >> > (
        nrow, ncol, input_data_a, input_data_b, output_data);
    return 0;
}

int DLGpuL1LossGradient(const DLArrayHandle input, const DLArrayHandle input1,const DLArrayHandle in_grad,
    DLArrayHandle output) {

    assert(input->ndim == input1->ndim);
    int count = 1;
    for (int i = 0; i < input->ndim; ++i) {
    assert(input->ndim == input1->ndim);
        count *= input->shape[i];
    }

    int  n=input->shape[0];
    const float* inputArr = (const float*)input->data;
    const float* inputArr1 = (const float*)input1->data;
    const float* gradArr = (const float*)in_grad->data;
    float* outputArr = (float*)output->data;
    int nrow=input->shape[0];
    dim3 threads;
    threads.x = nrow;

    matrix_l1lossgradient_kernel << <1, threads >> > (
        inputArr, inputArr1,gradArr, outputArr, count,n);
    return 0;
}
int DLGpuL2LossGradient(const DLArrayHandle input, const DLArrayHandle input1,const DLArrayHandle in_grad,
    DLArrayHandle output) {

    assert(input->ndim == input1->ndim);
    int count = 1;
    for (int i = 0; i < input->ndim; ++i) {
    assert(input->ndim == input1->ndim);
        count *= input->shape[i];
    }

    int  n=input->shape[0];
    const float* inputArr = (const float*)input->data;
    const float* inputArr1 = (const float*)input1->data;
    const float* gradArr = (const float*)in_grad->data;
    float* outputArr = (float*)output->data;
    int nrow=input->shape[0];
    dim3 threads;
    threads.x = nrow;

    matrix_l2lossgradient_kernel << <1, threads >> > (
        inputArr, inputArr1,gradArr, outputArr, count,n);
    return 0;
}
int DLGpuL1regular(const DLArrayHandle input_a,
    DLArrayHandle output) {
    assert(input_a->ndim == 2);
    assert(output->ndim == 1);
    int nrow = input_a->shape[0];
    assert(nrow <= 1024 * 4);
    int ncol = input_a->shape[1];
    const float* input_data_a = (const float*)input_a->data;
    float* output_data = (float*)output->data;
    dim3 threads;
    if (nrow <= 1024) {
        threads.x = nrow;
    }
    else {
        threads.x = 1024;
        threads.y = (nrow + 1023) / 1024;
    }
    // 1 block, each block with 'threads' number of threads with 'nrow' shared
    // memory size
    matrix_l1regular_kernel << <1, threads, nrow * sizeof(float) >> > (
        nrow, ncol, input_data_a, output_data);
    return 0;
}
int DLGpuL2regular(const DLArrayHandle input_a,
    DLArrayHandle output) {
    assert(input_a->ndim == 2);
    assert(output->ndim == 1);
    int nrow = input_a->shape[0];
    assert(nrow <= 1024 * 4);
    int ncol = input_a->shape[1];
    const float* input_data_a = (const float*)input_a->data;
    float* output_data = (float*)output->data;
    dim3 threads;
    if (nrow <= 1024) {
        threads.x = nrow;
    }
    else {
        threads.x = 1024;
        threads.y = (nrow + 1023) / 1024;
    }
    // 1 block, each block with 'threads' number of threads with 'nrow' shared
    // memory size
    matrix_l2regular_kernel << <1, threads, nrow * sizeof(float) >> > (
        nrow, ncol, input_data_a, output_data);
    return 0;
}

int DLGpuL1regularGradient(const DLArrayHandle input,const DLArrayHandle in_grad,
    DLArrayHandle output) {

    int count = 1;
    for (int i = 0; i < input->ndim; ++i) {
        count *= input->shape[i];
    }

    int  n=input->shape[0];
    const float* inputArr = (const float*)input->data;
    const float* gradArr = (const float*)in_grad->data;
    float* outputArr = (float*)output->data;
    int nrow=input->shape[0];
    dim3 threads;
    threads.x = nrow;

    matrix_l1regulargradient_kernel << <1, threads >> > (
        inputArr, gradArr, outputArr, count,n);
    return 0;
}
int DLGpuL2regularGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
    DLArrayHandle output) {

    int count = 1;
    for (int i = 0; i < input->ndim; ++i) {
        count *= input->shape[i];
    }

    int  n=input->shape[0];
    const float* inputArr = (const float*)input->data;
    const float* gradArr = (const float*)in_grad->data;
    float* outputArr = (float*)output->data;
    int nrow=input->shape[0];
    dim3 threads;
    threads.x = nrow;

    matrix_l2regulargradient_kernel << <1, threads >> > (
        inputArr, gradArr, outputArr, count,n);
    return 0;
}
















